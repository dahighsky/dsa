#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void vectorMul(int *A, int *B, int *C, int n){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(i<n and j<n){
		int sum = 0;
		for(int k=0;k<n;k++){
			sum+=A[i*n+k] * B[k*n+j];
		}
		C[i*n+j]=sum;
	}
}

int main(){
	int N;
	N = 512;
	
	int *h_A = new int[N*N];
	int *h_B = new int[N*N];
	int *h_C = new int[N*N];
	srand(time(0));
	for(int i=0;i<N*N;i++){
		h_A[i] = 1;
		h_B[i] = 1;
	}
	int *d_A, *d_B, *d_C;

	size_t size = N*N*sizeof(int);

	hipMalloc(&d_A,size);
	hipMalloc(&d_B,size);
	hipMalloc(&d_C,size);

	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);

	dim3 threadsperblock(16,16);
	dim3 blockspergrid((N+15)/16,(N+15)/16);

	vectorMul<<<blockspergrid,threadsperblock>>>(d_A,d_B,d_C,N);
	
	hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
	
	 // Print a small part of result
    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            cout << h_C[i * N + j] << " ";
        }
        cout << endl;
    }
}